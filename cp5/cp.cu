#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "cp.h"
using namespace std;
#include "../common/stopwatch.h"
ppc::stopwatch sw;


inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}


void preprocess(int ny, int nx, const float* data, float* aux) {

    for (int j = 0; j < ny; j++) {

        float sum;
        float avg;
        float e;

        sum = 0;

        for (int i = 0; i < nx; i++) {
            sum += data[j*nx+i];
        }

        avg = sum/nx;

        for (int i = 0; i < nx; i++) {
            aux[j*nx+i] = data[j*nx+i] - avg;
        }

        sum = 0;

        for (int i = 0; i < nx; i++) {
            e = aux[j*nx+i];
            sum += e*e;
        }

        sum = sqrt(sum);

        for (int i = 0; i < nx; i++) {
            aux[j*nx+i] /= sum;
        }
    }
}


__global__ void mykernel( int ny, int nx, float* data, float* result) 
{
	constexpr int A = 16;
    constexpr int B = 8;

	int jjj = threadIdx.y + blockIdx.y * blockDim.y;
    int iii = threadIdx.x + blockIdx.x * blockDim.x;

    int jj = jjj*A;
    int ii = iii*B;

    if (ii < jj) return;

    float block[A][B] = {0};

    for (int k = 0; k < nx; k++) {
        for (int m = 0; m < A; m++) {

        	float a = data[k*ny + (jj+m)];

            for (int l = 0; l < B; l++) {

                float b = data[k*ny + (ii+l)];
                block[m][l] += a*b;
            }
        }
    }
    
    int M = (jj+A) < ny ? A : ny-jj;
    int L = (ii+B) < ny ? B : ny-ii;

    for (int m = 0; m < M; m++) {
        for (int l = 0; l < L; l++) 
        {
            result[(jj+m)*ny+(ii+l)] = block[m][l];
        }
    }
}


__global__ void myppkernel(int ny, int nx, float* raw) 
{
    int j = threadIdx.x + blockIdx.y*blockDim.x;

    if (j < ny) {

        float sum;
        float avg;
        float e;

        sum = 0;

        for (int i = 0; i < nx; i++) {
            sum += raw[i*ny+j];
        }

        avg = sum/nx;

        for (int i = 0; i < nx; i++) {
            raw[i*ny+j] -= avg;
        }

        sum = 0;

        for (int i = 0; i < nx; i++) {
            e = raw[i*ny+j];
            sum += e*e;
        }

        sum = sqrt(sum);

        for (int i = 0; i < nx; i++) {
            raw[i*ny+j] /= sum;
        }
    }
}


__global__ void mytranskernel(int ny, int nx, float* raw, float* data) 
{
    int j = blockIdx.y;
    int ii = threadIdx.x;

    for (int i = ii; i < nx; i+=64) data[i*ny + j] = raw[j*nx + i];
}


void correlate(int ny, int nx, const float* data, float* result) {

    float* dataGPU = NULL;
    CHECK(hipMalloc((void**)&dataGPU, nx * ny * sizeof(float)));

    int dim = max(ny,nx);
    float* resultGPU = NULL;
    CHECK(hipMalloc((void**)&resultGPU, dim * dim * sizeof(float)));

    CHECK(hipMemcpy(resultGPU, data, ny * nx * sizeof(float), hipMemcpyHostToDevice));

	// Run transpose kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, ny);

        mytranskernel<<<dimGrid, dimBlock>>>(ny, nx, resultGPU, dataGPU);
        CHECK(hipGetLastError());
    }

    // Run preprocess kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, divup(ny, 64));

        myppkernel<<<dimGrid, dimBlock>>>(ny, nx, dataGPU);
        CHECK(hipGetLastError());
    }
    
    // Run multiplier kernel
    {
        dim3 dimBlock(8, 4);
        dim3 dimGrid(divup(ny, 64), divup(ny, 64));

        mykernel<<<dimGrid, dimBlock>>>(ny, nx, dataGPU, resultGPU);
        CHECK(hipGetLastError());
    }

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resultGPU));
}



