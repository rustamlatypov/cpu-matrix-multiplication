#include "hip/hip_runtime.h"
#include "is.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <omp.h>
#include <tuple>
#include <math.h>
#include <cmath>
#include "../common/stopwatch.h"
ppc::stopwatch sw;
using namespace std;

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}

void toFloat(int ny, int nx, const float* data, float* D) {

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++)
        {
            D[j*nx + i] = data[3*j*nx + 3*i + 0];
        }
    }
}

void process(int ny, int nx, float* D, float* S) {

    for (int j = 0; j < ny+1; j++) {
        S[j*(nx+1)] = 0;
    }

    for (int i = 0; i < nx+1; i++) {
        S[i] = 0;
    }

    for (int j = 1; j < ny+1; j++) {
        for (int i = 1; i < nx+1; i++) {

            float sum = S[(j-1)*(nx+1) + (i)];

            for (int ii = 0; ii < i; ii++) {
                sum += D[(j-1)*nx + ii];
            }

            S[j*(nx+1) + i] = sum;
        }
    }
}

__global__ void segkernel(int ny, int nx, float* S, float* resultGPU, int P, int vp, int vp2) 
{

	int X1 = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int X2 = threadIdx.x + blockIdx.x * blockDim.x + 1;

    if (X1 > ny || X2 > nx) return;

    //printf("(%d,%d)", X1, X2);

    float h = 0.0;
    float y0 = 0;
    float x0 = 0;

    float X = X1*X2;
    float Y = P - X;

    float A = (X+Y)/(X*Y);
    float B = -2*vp/Y;
    float C = vp2/Y;

    for (int j0 = 0; j0 <= ny-X1; j0++) {
        for (int i0 = 0; i0 <= nx-X2; i0++) {

            int j1 = j0+X1;
            int i1 = i0+X2;

            float vx = S[j1*(nx+1) + i1]
                       - S[j0*(nx+1) + i1]
                       - S[j1*(nx+1) + i0]
                       + S[j0*(nx+1) + i0];

            float h_value = (vx*A + B)*vx + C;

            //printf("(%d,%d,%d,%d,%.3f)", j0, i0, j1, i1, h_value);

            if (h_value > h)
            {
            	h = h_value;
            	y0 = j0;
			    x0 = i0;
            }
        }
    }

    //printf("\n%.3f, %d, %d, %d, %d", h, y0, x0, y1, x1);

    int ind = ((X1-1)*nx + (X2-1))*5;
    resultGPU[ind] = h;
    resultGPU[ind + 1] = y0;
    resultGPU[ind + 2] = x0;
    resultGPU[ind + 3] = y0+X1;
    resultGPU[ind + 4] = x0+X2;
} 


Result segment(int ny, int nx, const float* data) {

    vector<float> D(ny*nx);
    toFloat(ny, nx, data, D.data());

    vector<float> S((ny+1)*(nx+1));
    process(ny, nx, D.data(), S.data());

    //print(ny, nx, D);
    //print(ny+1, nx+1, S);

    float vp = 0;
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++)
        {
            vp += D[j*nx + i];
        }
    }
    float vp2 = vp*vp;
    float P = ny*nx;
   

    float* sGPU = NULL;
    CHECK(hipMalloc((void**)&sGPU, (ny+1) * (nx+1) * sizeof(float)));
    CHECK(hipMemcpy(sGPU, S.data(), (ny+1) * (nx+1) * sizeof(float), hipMemcpyHostToDevice));


	vector<float> result(5 * ny * nx);
    float* resultGPU = NULL;
    CHECK(hipMalloc((void**)&resultGPU, 5 * ny * nx * sizeof(float)));


    // Run kernel
    {
        dim3 dimBlock(16, 16);
        dim3 dimGrid(divup(nx, 16), divup(ny, 16));

        segkernel<<<dimGrid, dimBlock>>>(ny, nx, sGPU, resultGPU, P, vp, vp2);
        CHECK(hipGetLastError());
    }


    // Copy data back, release memory, find best solution
    CHECK(hipMemcpy(result.data(), resultGPU, 5 * ny * nx * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(sGPU));
    CHECK(hipFree(resultGPU));

    /*
    cout << endl;
    for (int i = 0; i < 5*ny*nx; i++)
    {
    	cout << result[i] << " ";
    }
    cout << endl;
    */


    float h = 0.0;
    int y0 = 0;
    int x0 = 0;
    int y1 = 0;
    int x1 = 0;

    for (int i = 0; i < 5*ny*nx; i+=5) {
    	if (result[i] > h)
    	{
    		h = result[i];
    		y0 = result[i+1];
    		x0 = result[i+2];
    		y1 = result[i+3];
    		x1 = result[i+4];
    	}
    }


    float a = 0;
    for (int j = y0; j < y1; j++) {
        for (int i = x0; i < x1; i++) {

            a += D[j*nx + i];

        }
    }

    float b = vp - a;

    float X = (y1-y0)*(x1-x0);
    a /= X;

    float Y = P - X;
    b /= Y;

    return { y0, x0, y1, x1, {(float) b, (float) b, (float) b},
                    		 {(float) a, (float) a, (float) a} };
}










