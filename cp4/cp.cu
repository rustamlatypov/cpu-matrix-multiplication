#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include "cp.h"
using namespace std;



inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)



inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}



static void print(int ny, int nx, const float *matrix) {
    for(int j = 0; j < ny; j++) {
        for(int i = 0; i < nx; i++) {
            float x = matrix[j*nx+i];
            if(std::abs(x) < 10.0) {
                printf("% -7.3f", x);
            } else {
                printf("% -7.0e", x);
            }
        }
        std::cout << '\n';
    }
    std::cout << '\n';
}



__global__ void mykernel(const float* data, float* result, int ny, int nx) 
{
	int j = threadIdx.y + blockIdx.y * blockDim.y;
    int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i <= j || j >= ny || i >= ny) return;

    double e = 0;

    for (int k = 0; k < nx; k++)
    {
        e += data[j*nx+k] * data[i*nx+k];
    }

    result[j*ny+i] = e;
}




void correlate(int ny, int nx, const float* data, float* result) {

    std::vector<float> aux(ny*nx);

    for (int i = 0; i < ny; i++) {

        float sum;
        float avg;
        float e;

        sum = 0;

        for (int j = 0; j < nx; j++) {
            sum += data[i*nx+j];
        }

        avg = sum/nx;

        for (int j = 0; j < nx; j++) {
            aux[i*nx+j] = data[i*nx+j] - avg;
        }

        sum = 0;

        for (int j = 0; j < nx; j++) {
            e = aux[i*nx+j];
            sum += e*e;
        }

        sum = sqrt(sum);

        for (int j = 0; j < nx; j++) {
            aux[i*nx+j] /= sum;
        }
    }

    float* dataGPU = NULL;
    CHECK(hipMalloc((void**)&dataGPU, ny * nx * sizeof(float)));

    float* resultGPU = NULL;
    CHECK(hipMalloc((void**)&resultGPU, ny * ny * sizeof(float)));

    CHECK(hipMemcpy(dataGPU, aux.data(), ny * nx * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(dataGPU, resultGPU, ny, nx);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(resultGPU));
    
	for (int i = 0; i < ny; i++) 
	{
        result[i*ny+i] = 1;
    }
}








